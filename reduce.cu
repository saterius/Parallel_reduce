
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define N 2048

//Interleave addressing kernel version
__global__ void interleaved_reduce(int* d_in, int* d_out) {

	int i = threadIdx.x;
	int M = N/2;
	__shared__ int sharedMem[N];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	sharedMem[i] = d_in[id];
	__syncthreads();
	for(int s = 1; s < N; s = s<<1) {
		if(i < M) {
			//printf("stride = %d, thread %d is active \n", s, i);
			sharedMem[(2*s)*id] = sharedMem[(2*s)*id] + sharedMem[(2*s)*id+s];
		}
		__syncthreads();
		M = M/2;
	}
	if(i == 0)
		d_out[0] = sharedMem[0];
}

//Contiguous addressing kernel version
__global__ void contiguous_reduce(int* d_in, int* d_out){

	int i = threadIdx.x;
	int M = N/2;
	__shared__ int sharedMem[N];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	sharedMem[i] = d_in[id];
	__syncthreads();
	for(int s = M; s > 0; s = s>>1) {
		if(i < M) {
			//printf("stride = %d, thread %d is active \n", s, i);
			sharedMem[id] = sharedMem[id] + sharedMem[id+s];
		}
		__syncthreads();
		M = M/2;
	}
	if(i == 0)
		d_out[0] = sharedMem[0];
}

int main()
{
	int h_in[N];
	int h_out = 0;

	for(int i = 0; i < N; i++)
		h_in[i] = i+1;

	int *d_in, *d_out;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**) &d_in, N*sizeof(int));
	hipMalloc((void**) &d_out, sizeof(int));
	hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start);

	//kernel call
	//interleaved_reduce<<<1, 1024>>>(d_in, d_out);
	contiguous_reduce<<<1, 1024>>>(d_in, d_out);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_in);
	hipFree(d_out);

	//printf("Output: %d\n", h_out);
	printf("%f\n", milliseconds);
	
	return -1;
}
